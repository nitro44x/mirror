#include "hip/hip_runtime.h"
#include "simt_macros.hpp"
#include "simt_allocator.hpp"
#include "simt_vector.hpp"

#include <vector>
#include <numeric>

#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

template <typename T>
__global__ void printArray(T const* data, size_t size) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printf("gpu v = ");
		for (int i = 0; i < size; ++i)
			printf("%lf ", data[i]);
		printf("\n");
	}
}

HOSTDEVICE void printVector(simt::containers::vector<double> const & v) {
	printf("gpu v = ");
	for (auto const& d : v)
		printf("%lf ", d);
	printf("\n");
}

__global__ void call_printVector(simt::containers::vector<double> const& v) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printVector(v);
	}
}

__global__ void call_printVector_ref(simt::containers::vector<double> const& v) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printVector(v);
	}
}


HOSTDEVICE void setTo(simt::containers::vector<double> & v, simt::containers::vector<double>::value_type value) {
	for (auto & d : v)
		d = value;
}

__global__ void call_setTo(simt::containers::vector<double> & v, simt::containers::vector<double>::value_type value) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		setTo(v, value);
	}
}


void test1() {
	std::cout << "std::vector" << std::endl;
	std::vector<double, simt::memory::managed_allocator<double>> v(10);
	std::iota(begin(v), end(v), -4);
	std::cout << "cpu v = ";
	for (auto const& d : v)
		std::cout << d << " ";
	std::cout << std::endl;

	printArray<<<1,1>>>(v.data(), v.size());
	simt_sync
	std::cout << std::endl;
}

void test2() {
	std::cout << "simt::containers::vector [raw ptr]" << std::endl;
	simt::containers::vector<double> simt_v(10, 3.0);
	std::iota(simt_v.begin(), simt_v.end(), -3.0);
	simt_v.push_back(4321);
	std::cout << "cpu v = ";
	for (auto const& d : simt_v)
		std::cout << d << " ";
	std::cout << std::endl;
	printArray<<<1,1>>>(simt_v.data(), simt_v.size());
	simt_sync
	std::cout << std::endl;
}

void test3() {
	std::cout << "simt::containers::vector [object]" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>(10);
	std::iota(simt_v_ptr->begin(), simt_v_ptr->end(), -4);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	printVector(*simt_v_ptr);
	call_printVector<<<1,1>>>(*simt_v_ptr);
	simt_sync
	delete simt_v_ptr;
	std::cout << std::endl;
}

void test3a() {
	std::cout << "simt::containers::vector [object] printByRef" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>(10);
	std::iota(simt_v_ptr->begin(), simt_v_ptr->end(), -4);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	printVector(*simt_v_ptr);
	call_printVector_ref<<<1,1>>>(*simt_v_ptr);
	simt_sync
	delete simt_v_ptr;
	std::cout << std::endl;
}

void test4() {
	std::cout << "modify simt::containers::vector [object] on cpu" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>;
	simt_v_ptr->resize(10);
	setTo(*simt_v_ptr, 123);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	call_printVector<<<1,1>>>(*simt_v_ptr);
	simt_sync
	delete simt_v_ptr;
	simt_sync
	std::cout << std::endl;
}

void test5() {
	std::cout << "modify simt::containers::vector [object] on gpu" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>;
	simt_v_ptr->resize(10);
	call_setTo<<<1,1>>>(*simt_v_ptr, 123);
	simt_sync
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	call_printVector << <1, 1 >> > (*simt_v_ptr);
	simt_sync
	delete simt_v_ptr;
	std::cout << std::endl;
}

void test6() {
	std::cout << "modify simt::containers::vector [object] on gpu" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>;
	for (auto i = 0; i < 4; ++i)
		simt_v_ptr->push_back(10 + i * i);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	call_printVector<<<1,1>>>(*simt_v_ptr);
	simt_sync
	delete simt_v_ptr;
	std::cout << std::endl;
}


enum class ABC_t { Unk, A, B, C };

struct encodedObj {
	ABC_t type;
	double d = 0;
	int i = 0;
};

class A {
public:
	HOSTDEVICE virtual ~A() { ; }

	HOSTDEVICE virtual void sayHi() = 0;

	HOSTDEVICE virtual encodedObj encode() const = 0;
	HOSTDEVICE virtual void decode(encodedObj e) = 0;
	HOSTDEVICE virtual ABC_t type() const = 0;
};

class B : public A {
public:
	HOSTDEVICE B() { ; }
	HOSTDEVICE B(int j) : j(j) {}
	HOSTDEVICE B(encodedObj e) : B() { decode(e); }
	HOSTDEVICE ~B() override { ; }

	HOSTDEVICE void sayHi() override {
		//printf("Hello from B, j = %d\n", j);
		++j;
	}

	HOSTDEVICE virtual encodedObj encode() const {
		return { ABC_t::B, 0, j };
	}

	HOSTDEVICE virtual void decode(encodedObj e) {
		j = e.i;
	}

	HOSTDEVICE virtual ABC_t type() const {
		return ABC_t::B;
	}

	int j = 0;
};

class C : public A {
public:
	HOSTDEVICE C() { ; }
	HOSTDEVICE C(int j) : d(j) {}
	HOSTDEVICE C(encodedObj e) : C() { decode(e); }

	HOSTDEVICE ~C() override { ; }

	HOSTDEVICE void sayHi() override {
		//printf("Hello from C, d = %lf\n", d);
		++d;
	}

	HOSTDEVICE virtual encodedObj encode() const {
		return { ABC_t::C, d, 0 };
	}

	HOSTDEVICE virtual void decode(encodedObj e) {
		d = e.d;
	}

	HOSTDEVICE virtual ABC_t type() const {
		return ABC_t::C;
	}

	double d = 0;
};

__global__
void allocateDeviceObjs(simt::containers::vector<A*> & device_objs, simt::containers::vector<encodedObj> const& encoded_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (; tid < encoded_objs.size(); tid += blockDim.x * gridDim.x) {
		switch (encoded_objs[tid].type) {
		case ABC_t::B:
			//printf("Allocating B object! %d\n", (int)tid);
			device_objs[tid] = new B(encoded_objs[tid]);
			break;
		case ABC_t::C:
			//printf("Allocating C object! %d\n", (int)tid);
			device_objs[tid] = new C(encoded_objs[tid]);
			break;
		case ABC_t::A:
		case ABC_t::Unk:
		default:
			printf("Error allocating object!\n");
		}

		if (nullptr == device_objs[tid])
			printf("failed allocation at tid = %u\n", tid);
	}
}

__global__
void sayHi(simt::containers::vector<A*> & device_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (; tid < device_objs.size(); tid += blockDim.x * gridDim.x) {
		//printf("Saying hi from an A* \n");
		device_objs[tid]->sayHi();
	}
}

__global__
void deallocateDeviceObjs(simt::containers::vector<A*> & device_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (; tid < device_objs.size(); tid += blockDim.x * gridDim.x) {
		//printf("Deallocating an A*\n");
		delete device_objs[tid];
	}
}

template <typename T>
__global__ void compute_sizeof(size_t * size) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid == 0)
		*size = sizeof(T);
}

template <typename T>
HOST size_t getDeviceSize() {
	size_t * size = nullptr;
	hipMallocManaged((void**)&size, sizeof(size_t));
	compute_sizeof<T><<<1,1>>>(size);
	simt_sync;
	auto const result = *size;
	hipFree(size);
	return result;
}

void test7() {
	const auto N = 10;
	std::vector<A*> host_objs;
	auto encoded_objs = new simt::containers::vector<encodedObj>();
	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new B(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}

	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new C(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}

	auto const nBlocks = 128;
	auto const nThreadsPerBlock = 128;
	auto device_objs = new simt::containers::vector<A*>(encoded_objs->size(), nullptr);
	allocateDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs, *encoded_objs);
	simt_sync
	delete encoded_objs;

	size_t nNulls = 0;
	for (auto const& p : *device_objs) {
		if (p == nullptr)
			++nNulls;
	}

	if (nNulls > 0) {
		std::cout << "Found " << nNulls << " nullptrs" << std::endl;
		return;
	}

	sayHi<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	simt_sync
	deallocateDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	simt_sync

	delete device_objs;
	for (auto o : host_objs)
		delete o;
}

void test8() {
	thrust::device_vector<double> d;
	d.resize(10);
	for (size_t i = 0; i < d.size(); ++i)
		d[i] = (double)i;

	thrust::host_vector<double> h = d;
	for (auto const& i : h)
		std::cout << i << " ";
	std::cout << std::endl;

}

void test9() {
	std::cout << "cpu sizeof(A) = " << sizeof(A) << std::endl;
	std::cout << "gpu sizeof(A) = " << getDeviceSize<A>() << std::endl;
	std::cout << "cpu sizeof(B) = " << sizeof(B) << std::endl;
	std::cout << "gpu sizeof(B) = " << getDeviceSize<B>() << std::endl;
	std::cout << "cpu sizeof(C) = " << sizeof(C) << std::endl;
	std::cout << "gpu sizeof(C) = " << getDeviceSize<C>() << std::endl;
}

__global__
void constructDeviceObjs(simt::containers::vector<A*> & device_objs, simt::containers::vector<encodedObj> const& encoded_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;

	for (; tid < encoded_objs.size(); tid += blockDim.x * gridDim.x) {

		switch (encoded_objs[tid].type) {
		case ABC_t::B:
			//printf("constructing B object at %p \n", device_objs[tid]);
			new(device_objs[tid]) B(encoded_objs[tid]);
			break;
		case ABC_t::C:
			//printf("constructing C object at %p \n", device_objs[tid]);
			new(device_objs[tid]) C(encoded_objs[tid]);
			break;
		case ABC_t::A:
		case ABC_t::Unk:
		default:
			printf("Error allocating object!\n");
		}

		if (nullptr == device_objs[tid])
			printf("failed allocation at tid = %u\n", tid);
	}
}

__global__
void destructDeviceObjs(simt::containers::vector<A*> & device_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (; tid < device_objs.size(); tid += blockDim.x * gridDim.x) {
		//printf("Deallocating an A*\n");
		device_objs[tid]->~A();
	}
}

void test10() {
	const auto N = 5;
	std::vector<A*> host_objs;
	auto encoded_objs = new simt::containers::vector<encodedObj>();
	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new B(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}

	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new C(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}

	auto const nBlocks = 128;
	auto const nThreadsPerBlock = 128;
	auto device_objs = new simt::containers::vector<A*>(encoded_objs->size(), nullptr);
	auto const sizeofB = getDeviceSize<B>();
	auto const sizeofC = getDeviceSize<C>();
	for(size_t i = 0; i < encoded_objs->size(); ++i)
		hipMallocManaged((void**)&(*device_objs)[i], (*encoded_objs)[i].type == ABC_t::B ? sizeofB : sizeofC);

	constructDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs, *encoded_objs);
	simt_sync
	

	sayHi<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	simt_sync
	destructDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	simt_sync


	delete encoded_objs;
	for (auto p : *device_objs)
		hipFree(p);
	delete device_objs;

	for (auto o : host_objs)
		delete o;
}

void test11() {
	const auto N = 5;
	std::vector<A*> host_objs;
	auto encoded_objs = new simt::containers::vector<encodedObj>();
	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new B(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}

	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new C(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}

	auto const nBlocks = 128;
	auto const nThreadsPerBlock = 128;
	auto device_objs = new simt::containers::vector<A*>(encoded_objs->size(), nullptr);
	auto const sizeofB = getDeviceSize<B>();
	auto const sizeofC = getDeviceSize<C>();

	auto sizeofFold = [sizeofB, sizeofC](size_t currentTotal, encodedObj const& e) {
		switch (e.type) {
		case ABC_t::B:
			return currentTotal + sizeofB;
		case ABC_t::C:
			return currentTotal + sizeofC;
		default:
			assert(false);
			return size_t(0);
		}
	};

	auto totalSpaceNeeded_bytes = std::accumulate(encoded_objs->begin(), encoded_objs->end(), size_t(0), sizeofFold);
	std::cout << "total Space needed [bytes] = " << totalSpaceNeeded_bytes << std::endl;

	auto tank = new simt::containers::vector<char, simt::memory::device_allocator<char>>(totalSpaceNeeded_bytes, '\0');

	std::cout << "               Tank setup" << std::endl;
	std::cout << "--------------------------" << std::endl;

	size_t offset = 0;
	for (size_t i = 0; i < encoded_objs->size(); ++i) {
		
		(*device_objs)[i] = (A*)(tank->data() + offset);

		auto const& e = (*encoded_objs)[i];
		switch (e.type) {
		case ABC_t::B:
			offset += sizeofB;
			break;
		case ABC_t::C:
			offset += sizeofC;
			break;
		default:
			assert(false);
		}
	}

	for (size_t i = 0; i < device_objs->size(); ++i) {
		if(i < 3 || i+3 > device_objs->size())
			std::cout << "A[" << i << "] = " << (*device_objs)[i] << std::endl;
	}

	auto const tankStart = &(*(tank->begin()));
	auto const tankEnd = &(*(--(tank->end()))) + sizeof(char);
	std::cout << "tank start = " << (void*)tankStart << std::endl;
	std::cout << "tank end   = " << (void*)tankEnd << std::endl;

	constructDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs, *encoded_objs);
	simt_sync

	for(size_t i = 0; i < 10000; ++i)
		sayHi<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	std::cout << "Launched a bunch of sayHi's" << std::endl;
	simt_sync
	destructDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	simt_sync


	delete encoded_objs;
	delete tank;
	delete device_objs;

	for (auto o : host_objs)
		delete o;
}

void test12() {

}
