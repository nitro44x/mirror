#include "hip/hip_runtime.h"
#include "simt_macros.hpp"
#include "simt_allocator.hpp"
#include "simt_vector.hpp"

#include <vector>
#include <numeric>

#include <iostream>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <>

template <typename T>
__global__ void printArray(T const* data, size_t size) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printf("gpu v = ");
		for (int i = 0; i < size; ++i)
			printf("%lf ", data[i]);
		printf("\n");
	}
}

HOSTDEVICE void printVector(simt::containers::vector<double> const & v) {
	printf("gpu v = ");
	for (auto const& d : v)
		printf("%lf ", d);
	printf("\n");
}

__global__ void call_printVector(simt::containers::vector<double> const& v) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printVector(v);
	}
}

__global__ void call_printVector_ref(simt::containers::vector<double> const& v) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		printVector(v);
	}
}


HOSTDEVICE void setTo(simt::containers::vector<double> & v, simt::containers::vector<double>::value_type value) {
	for (auto & d : v)
		d = value;
}

__global__ void call_setTo(simt::containers::vector<double> & v, simt::containers::vector<double>::value_type value) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		setTo(v, value);
	}
}


void test1() {
	std::cout << "std::vector" << std::endl;
	std::vector<double, simt::memory::managed_allocator<double>> v(10);
	std::iota(begin(v), end(v), -4);
	std::cout << "cpu v = ";
	for (auto const& d : v)
		std::cout << d << " ";
	std::cout << std::endl;

	printArray << <1, 1 >> > (v.data(), v.size());
	hipDeviceSynchronize();
	std::cout << std::endl;
}

void test2() {
	std::cout << "simt::containers::vector [raw ptr]" << std::endl;
	simt::containers::vector<double> simt_v(10, 3.0);
	std::iota(simt_v.begin(), simt_v.end(), -3.0);
	simt_v.push_back(4321);
	std::cout << "cpu v = ";
	for (auto const& d : simt_v)
		std::cout << d << " ";
	std::cout << std::endl;
	printArray << <1, 1 >> > (simt_v.data(), simt_v.size());
	hipDeviceSynchronize();
	std::cout << std::endl;
}

void test3() {
	std::cout << "simt::containers::vector [object]" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>(10);
	std::iota(simt_v_ptr->begin(), simt_v_ptr->end(), -4);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	printVector(*simt_v_ptr);
	call_printVector << <1, 1 >> > (*simt_v_ptr);
	hipDeviceSynchronize();
	delete simt_v_ptr;
	std::cout << std::endl;
}

void test3a() {
	std::cout << "simt::containers::vector [object] printByRef" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>(10);
	std::iota(simt_v_ptr->begin(), simt_v_ptr->end(), -4);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	printVector(*simt_v_ptr);
	call_printVector_ref << <1, 1 >> > (*simt_v_ptr);
	hipDeviceSynchronize();
	delete simt_v_ptr;
	std::cout << std::endl;
}

void test4() {
	std::cout << "modify simt::containers::vector [object] on cpu" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>;
	simt_v_ptr->resize(10);
	setTo(*simt_v_ptr, 123);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	call_printVector << <1, 1 >> > (*simt_v_ptr);
	hipDeviceSynchronize();
	delete simt_v_ptr;
	std::cout << std::endl;
}

void test5() {
	std::cout << "modify simt::containers::vector [object] on gpu" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>;
	simt_v_ptr->resize(10);
	call_setTo << <1, 1 >> > (*simt_v_ptr, 123);
	hipDeviceSynchronize();
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	call_printVector << <1, 1 >> > (*simt_v_ptr);
	hipDeviceSynchronize();
	delete simt_v_ptr;
	std::cout << std::endl;
}

void test6() {
	std::cout << "modify simt::containers::vector [object] on gpu" << std::endl;
	auto simt_v_ptr = new simt::containers::vector<double>;
	for (auto i = 0; i < 4; ++i)
		simt_v_ptr->push_back(10 + i * i);
	std::cout << "cpu v = ";
	for (auto const& d : *simt_v_ptr)
		std::cout << d << " ";
	std::cout << std::endl;
	call_printVector << <1, 1 >> > (*simt_v_ptr);
	hipDeviceSynchronize();
	delete simt_v_ptr;
	std::cout << std::endl;
}


enum class ABC_t { Unk, A, B, C };

struct encodedObj {
	ABC_t type;
	double d = 0;
	int i = 0;
};

class A {
public:
	HOSTDEVICE virtual ~A() { ; }

	HOSTDEVICE virtual void sayHi() = 0;

	HOSTDEVICE virtual encodedObj encode() const = 0;
	HOSTDEVICE virtual void decode(encodedObj e) = 0;
};

class B : public A {
public:
	HOSTDEVICE B() { ; }
	HOSTDEVICE B(int j) : j(j) {}
	HOSTDEVICE B(encodedObj e) : B() { decode(e); }
	HOSTDEVICE ~B() override { ; }

	HOSTDEVICE void sayHi() override {
		//printf("Hello from B, j = %d\n", j);
		++j;
	}

	HOSTDEVICE virtual encodedObj encode() const {
		return { ABC_t::B, 0, j };
	}

	HOSTDEVICE virtual void decode(encodedObj e) {
		j = e.i;
	}

	int j = 0;
};

class C : public A {
public:
	HOSTDEVICE C() { ; }
	HOSTDEVICE C(int j) : d(j) {}
	HOSTDEVICE C(encodedObj e) : C() { decode(e); }
	HOSTDEVICE ~C() override { ; }

	HOSTDEVICE void sayHi() override {
		//printf("Hello from C, d = %lf\n", d);
		++d;
	}

	HOSTDEVICE virtual encodedObj encode() const {
		return { ABC_t::C, d, 0 };
	}

	HOSTDEVICE virtual void decode(encodedObj e) {
		d = e.d;
	}

	double d = 0;
};

__global__
void allocateDeviceObjs(simt::containers::vector<A*> & device_objs, simt::containers::vector<encodedObj> const& encoded_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid == 0) 
		printf("device_objs size = %d\n", (int)device_objs.size());

	bool isRoot = tid == 0;

	for (; tid < encoded_objs.size(); tid += blockDim.x * gridDim.x) {
		if (isRoot)
			printf("tid = %d\n", (int)tid);
		switch (encoded_objs[tid].type) {
		case ABC_t::B:
			//printf("Allocating B object! %d\n", (int)tid);
			device_objs[tid] = new B(encoded_objs[tid]);
			break;
		case ABC_t::C:
			//printf("Allocating C object! %d\n", (int)tid);
			device_objs[tid] = new C(encoded_objs[tid]);
			break;
		case ABC_t::A:
		case ABC_t::Unk:
		default:
			printf("Error allocating object!\n");
		}
	}
}

__global__
void sayHi(simt::containers::vector<A*> & device_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (; tid < device_objs.size(); tid += blockDim.x * gridDim.x) {
		//printf("Saying hi from an A* \n");
		device_objs[tid]->sayHi();
	}
}

__global__
void deallocateDeviceObjs(simt::containers::vector<A*> & device_objs) {
	auto tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (; tid < device_objs.size(); tid += blockDim.x * gridDim.x) {
		//printf("Deallocating an A*\n");
		delete device_objs[tid];
	}
}

void test7() {
	const auto N = 6000;
	std::vector<A*> host_objs;
	auto encoded_objs = new simt::containers::vector<encodedObj>();
	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new B(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}

	for (auto i = 0; i < N; ++i) {
		host_objs.push_back(new C(i));
		encoded_objs->push_back(host_objs.back()->encode());
	}


	auto const nBlocks = 128;
	auto const nThreadsPerBlock = 128;
	auto device_objs = new simt::containers::vector<A*>(encoded_objs->size(), nullptr);
	allocateDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs, *encoded_objs);
	check(hipDeviceSynchronize());
	delete encoded_objs;

	for(size_t i = 0; i < 1000; ++i)
		sayHi<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	check(hipDeviceSynchronize());
	deallocateDeviceObjs<<<nBlocks,nThreadsPerBlock>>>(*device_objs);
	check(hipDeviceSynchronize());

	delete device_objs;
	for (auto o : host_objs)
		delete o;
}